#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void matrixMul (int *A_gpu,int *B_gpu,int *C_gpu,int N) {
	int k, accu = 0,i,j;
	i = blockIdx.y * blockDim.y + threadIdx.y;
	j = blockIdx.x * blockDim.x + threadIdx.x;

	if(i<N && j<N){
		for(k=0; k < N-1; k++) {
			accu += A_gpu[k*N+i] * B_gpu[k*N+j];
		}
	}

	C_gpu[i*N+j] = accu;
}

int main(int argc, char** argv) {
	int N;
	if(argc!=2) {
		printf("Invalid arguments");
		return -1;
	}
	
	N = atoi(argv[1]);

	int *A_cpu=(int *)malloc(N*N*sizeof(int)), *B_cpu=(int *)malloc(N*N*sizeof(int)) , *C_cpu=(int *)malloc(N*N*sizeof(int));
	int *A_gpu, *B_gpu, *C_gpu;
	int size= N*N*sizeof(int);
	int i,j,temp;
	hipEvent_t start, stop;
	float time;

	for(i=0; i<N; i++){
		for(j=0;j<N;j++){
			*(A_cpu+i*N+j) = 1;
			*(B_cpu+i*N+j) = 2;
		}
	}

	for(i=0;i<N;i++){
		for(j=0;j<N;j++){
			temp = *(A_cpu+i*N+j);
			*(A_cpu+i*N+j) = *(A_cpu+j*N+i);
			*(A_cpu+j*N+i) = temp;
		}
	}

	hipMalloc((void **)&A_gpu, size);
	hipMalloc((void **)&B_gpu, size);
	hipMalloc((void **)&C_gpu, size);

	A_cpu = (int *)malloc(size);
	B_cpu = (int *)malloc(size);
	C_cpu = (int *)malloc(size);	

	dim3 dimBlock(16,16);
	dim3 dimGrid((N+15)/16, (N+15)/16);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(A_gpu, A_cpu, size, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B_cpu, size, hipMemcpyHostToDevice);

	hipEventRecord(start,0);
	matrixMul<<<dimGrid, dimBlock>>>(A_gpu, B_gpu, C_gpu, N);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(C_cpu, C_gpu, size, hipMemcpyDeviceToHost);

	free(A_cpu);
	free(B_cpu);
	free(C_cpu);
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);

	printf("time for %d (excluding memcpy)= %f\n",N,time);
	return 0;
}

