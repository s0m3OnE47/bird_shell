
#include <hip/hip_runtime.h>
# include <stdio.h>
# include<stdlib.h>
# define N 10000
# define BLOCK_SIZE 16
# define SHARED_MEM (BLOCK_SIZE*BLOCK_SIZE)



__global__ void MatMul(int *A, int *B,int*C, int MatDim) {

    __shared__ int tempA[SHARED_MEM];
    __shared__ int tempB[SHARED_MEM];

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by*blockDim.y + threadIdx.y;
    int col = bx*blockDim.x + threadIdx.x;

    int temp =0;

    for(int i=0; i< (MatDim + BLOCK_SIZE - 1)/BLOCK_SIZE; ++i){

      if(row < MatDim  && (i*BLOCK_SIZE + ty < MatDim) ){
        tempA[ty*BLOCK_SIZE + tx] = A[i*MatDim*BLOCK_SIZE + ty*MatDim + row];
      }
      else
        tempA[ty*BLOCK_SIZE + tx] = 0;

      if(col < MatDim  && (i*BLOCK_SIZE + ty < MatDim)){
        tempB[ty*BLOCK_SIZE + tx] = B[i*MatDim*BLOCK_SIZE + ty*MatDim + col];
      }
      else
        tempB[ty*BLOCK_SIZE + tx] = 0;

      __syncthreads();

      for(int k=0;k<BLOCK_SIZE;++k){
        temp += tempA[ty*BLOCK_SIZE +k] * tempB[k*BLOCK_SIZE + tx];
      }
      __syncthreads();
    }

    if(row<MatDim && col<MatDim)
      C[row*MatDim + col] = temp;
}

int A[N][N], B[N][N],C[N][N], AT[N][N];

int main(){
  int *d_A, *d_B, *d_C;
  int size = N*N*sizeof(int);
  dim3 threads_per_block(BLOCK_SIZE,BLOCK_SIZE);
  dim3 blocks_in_grid((N+BLOCK_SIZE-1)/BLOCK_SIZE,(N+BLOCK_SIZE-1)/BLOCK_SIZE);
  hipEvent_t start, stop;
  float time;
 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for(int i=0;i<N;++i)
    for(int j=0;j<N;++j)
      A[i][j] = 5;
    for(int i=0;i<N;++i)
    for(int j=0;j<N;++j)
      B[i][j] = 5;
    for(int i=0;i<N;++i)
    for(int j=0;j<N;++j)
      AT[i][j] = A[j][i];
    hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_B,size);
    hipMalloc((void**)&d_C,size);
    hipEventRecord( start, 0 );



    hipMemcpy(d_A,AT,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

    MatMul<<<blocks_in_grid,threads_per_block>>>(d_A,d_B,d_C,N);
    hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );

    hipEventElapsedTime( &time, start, stop );




    hipEventDestroy( start );
    hipEventDestroy( stop );
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
  

  printf("Elapsed time is: %f\n",time);
return 0;

}
