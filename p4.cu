
#include <hip/hip_runtime.h>
# include <stdio.h>
# include<stdlib.h>
# define N 100
# define LOOP 1
# define BLOCK_SIZE 16
# define SHARED_MEM (BLOCK_SIZE*BLOCK_SIZE)



__global__ void MatMul(int *A, int *B,int*C, int MatDim) {

    __shared__ int tempA[SHARED_MEM];
    __shared__ int tempB[SHARED_MEM];

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by*blockDim.y + threadIdx.y;
    int col = bx*blockDim.x + threadIdx.x;

    int temp =0;

    for(int i=0; i< (MatDim + BLOCK_SIZE - 1)/BLOCK_SIZE; ++i){

      if(row < MatDim  && (i*BLOCK_SIZE + ty < MatDim) ){
        tempA[ty*BLOCK_SIZE + tx] = A[i*MatDim*BLOCK_SIZE + ty*MatDim + row];
      }
      else
        tempA[ty*BLOCK_SIZE + tx] = 0;

      if(col < MatDim  && (i*BLOCK_SIZE + ty < MatDim)){
        tempB[ty*BLOCK_SIZE + tx] = B[i*MatDim*BLOCK_SIZE + ty*MatDim + col];
      }
      else
        tempB[ty*BLOCK_SIZE + tx] = 0;

      __syncthreads();

      for(int k=0;k<BLOCK_SIZE;++k){
        temp += tempA[ty*BLOCK_SIZE +k] * tempB[k*BLOCK_SIZE + tx];
      }
      __syncthreads();
    }

    if(row<MatDim && col<MatDim)
      C[row*MatDim + col] = temp;
}

int A[N][N], B[N][N],C[N][N], AT[N][N];

int main(){
  int *d_A, *d_B, *d_C;
  int size = N*N*sizeof(int);
  dim3 threads_per_block(BLOCK_SIZE,BLOCK_SIZE);
  dim3 blocks_in_grid((N+BLOCK_SIZE-1)/BLOCK_SIZE,(N+BLOCK_SIZE-1)/BLOCK_SIZE);
  hipEvent_t start, stop;
  float time[LOOP];
  for(int k=0;k<LOOP;++k){
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for(int i=0;i<N;++i)
    for(int j=0;j<N;++j)
      A[i][j] = rand()%N;
    for(int i=0;i<N;++i)
    for(int j=0;j<N;++j)
      B[i][j] = rand()%N;
    for(int i=0;i<N;++i)
    for(int j=0;j<N;++j)
      AT[i][j] = A[j][i];
    hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_B,size);
    hipMalloc((void**)&d_C,size);
    hipEventRecord( start, 0 );



    hipMemcpy(d_A,AT,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

    hipEventRecord( start, 0 );
    MatMul<<<blocks_in_grid,threads_per_block>>>(d_A,d_B,d_C,N);
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );

    hipEventElapsedTime( &time[k], start, stop );

    hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);



    hipEventDestroy( start );
    hipEventDestroy( stop );
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
  }

float average =0;
for(int i=0;i<LOOP;++i)
  average = average + time[i];
average = average/LOOP;
  printf("Elapsed time is: %f\n",average);
return 0;

}
